#include <time.h>

#include <mma.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <assert.h>
#include <stdio.h>
#include <omp.h>

static inline void expAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    printf("assert failed: %s %s %d\n", hipGetErrorString(code), file, line);
    exit(code);
  }
}

static inline void expAssert(hipblasStatus_t status, const char *file, int line) {
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("assert failed, cublas error: status(%d) %s %d\n", status, file, line);
    exit(status);
  }
}

#define expErrChk(err) { expAssert((err), __FILE__, __LINE__); }

static float randf() {
    const int upper = 10;
    const int lower = -10;

    return ((rand() % (upper - lower + 1)) + lower) / 10.0;
}

static void randff(float *x, size_t size) {
    for (size_t i = 0; i < size; i++) {
        // printf("Setting %zu\n", i);
        x[i] = randf();
    }
}

static void init_mat(float *mat, int m, int n) {
  randff(mat, m * n);
}

static void copy_mat(half *d, const float *s, size_t size) {
  for (size_t idx = 0; idx < size; idx++) {
    d[idx] = s[idx];
  }
}

__global__ static void mat_mul_d(float *a, float *b, float *c,
                                 size_t m, size_t n, size_t k) {

  size_t idxX = threadIdx.x + (blockIdx.x * blockDim.x);
  size_t idxY = threadIdx.y + (blockIdx.y * blockDim.y);

  if (idxX >= n || idxY >= m) {
    return;
  }

  float sum = 0;

  for (size_t p = 0; p < k; p++) {
    sum += a[idxY * k + p] * b[p * n + idxX];
  }

  c[idxY * n + idxX] += sum;
}

static inline void mat_mul(float *da, float *db, float *dc,
                           size_t m, size_t n, size_t k) {

  int threadsY = 32;
  int threadsX = 32;
  int blocksY = (m + threadsY - 1) / threadsY;
  int blocksX = (n + threadsX - 1) / threadsX;

  dim3 threads(threadsX, threadsY);
  dim3 blocks(blocksX, blocksY);

  mat_mul_d<<<blocks, threads>>>(da, db, dc, m, n, k);
  expErrChk(hipGetLastError());
}

template<int threads, int scalingFactor>
__global__ static void mat_mul_dt(const half *a, const half *b, half *c,
                                  size_t m, size_t n, size_t k) {

  // each thread can process 16 * 16 elements
  // So, the cache should be able to feed all of the
  // threads simultaneously.
  // Cache, the bigger, the better.

  // ideally scaling factor should be = threads
  // that way each of the thread can work on it's own
  // slice of the matrix
  // But, that would also make the matrices not fit in the
  // shared memory.
  constexpr int tDim = 16;
  constexpr int sDim = tDim * scalingFactor;

  // sDim can be increased in size by using
  // co-operative thread groups?
  assert(threads % sDim == 0);

  __shared__ half aS[sDim * sDim];
  __shared__ half bS[sDim * sDim];
  __shared__ half cS[sDim * sDim];

  // j is the horizontal dimension
  // x is also the horizontal dimension
  const int jT = threadIdx.x;
  const int jB = blockIdx.x;
  const int iB = blockIdx.y;

  // the max warp index
  const int wMax = threads / warpSize;
  const int jW = jT / warpSize;
  const int jWT = jT % warpSize;

  // printf("jW: %d, scalingFactor: %d\n", jW, scalingFactor);

  assert(jW < wMax);

  assert((scalingFactor * scalingFactor) % wMax == 0);
  assert(sDim % warpSize == 0);
  assert(sDim % wMax == 0);
  assert(warpSize % tDim == 0);

  const int oFactor = wMax;
  const int maxBeats = sDim / warpSize;

  // load c
  for (int o = 0; o < sDim; o += oFactor) {

    for (int beat = 0; beat < maxBeats; beat++) {
      const int jIdx = jWT + (beat * warpSize);
      const int iIdx = jW + o;

      const int jIdxInBlock = jIdx % tDim;
      const int jBlockIdx = jIdx / tDim;
      const int iIdxInBlock = iIdx % tDim;
      const int iBlockIdx = iIdx / tDim;

      const int blockIdx = jBlockIdx + (iBlockIdx * scalingFactor);

      cS[jIdxInBlock + (iIdxInBlock * tDim) + (blockIdx * tDim * tDim)] = c[jIdx + (jB * sDim) + ((iIdx + (iB * sDim)) * n)];
    }
  }

  __syncthreads();

  for (size_t p0 = 0; p0 < k; p0 += sDim) {

    // load a
    for (int o = 0; o < sDim; o += oFactor) {
      for (int beat = 0; beat < maxBeats; beat++) {
        const int jIdx = jWT + (beat * warpSize);
        const int iIdx = jW + o;

        const int jIdxInBlock = jIdx % tDim;
        const int jBlockIdx = jIdx / tDim;
        const int iIdxInBlock = iIdx % tDim;
        const int iBlockIdx = iIdx / tDim;

        const int blockIdx = jBlockIdx + (iBlockIdx * scalingFactor);

        // actually, we can load to registers and use warp intrinsics
        // to shuffle the 16 halfs to the next warp
        // and store them all while avoiding any kinds of bank conflicts

        aS[jIdxInBlock + (iIdxInBlock * tDim) + (blockIdx * tDim * tDim)] = a[jIdx + p0 + ((iIdx + (iB * sDim)) * k)];
      }
    }

    // load b
    for (int o = 0; o < sDim; o += oFactor) {
      for (int beat = 0; beat < maxBeats; beat++) {
        const int jIdx = jWT + (beat * warpSize);
        const int iIdx = jW + o;

        const int jIdxInBlock = jIdx % tDim;
        const int jBlockIdx = jIdx / tDim;
        const int iIdxInBlock = iIdx % tDim;
        const int iBlockIdx = iIdx / tDim;

        const int blockIdx = jBlockIdx + (iBlockIdx * scalingFactor);

        bS[jIdxInBlock + (iIdxInBlock * tDim) + (blockIdx * tDim * tDim)] = b[jIdx + (jB * sDim) + ((iIdx + p0) * n)];
      }
    }

    __syncthreads();

    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, tDim, tDim, tDim, half, nvcuda::wmma::row_major> a_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, tDim, tDim, tDim, half, nvcuda::wmma::row_major> b_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, tDim, tDim, tDim, half> c_frag;

    assert(wMax == scalingFactor * scalingFactor);

    const int j = jW % scalingFactor;
    const int i = jW / scalingFactor;

    nvcuda::wmma::load_matrix_sync(c_frag, &cS[(j + (i * scalingFactor)) * tDim * tDim], tDim, nvcuda::wmma::mem_row_major);

    #pragma unroll
    for (int p = 0; p < scalingFactor; p++) {
      // Load the inputs
      nvcuda::wmma::load_matrix_sync(a_frag, &aS[(p + (i * scalingFactor)) * tDim * tDim], tDim);
      nvcuda::wmma::load_matrix_sync(b_frag, &bS[(j + (p * scalingFactor)) * tDim * tDim], tDim);

      // Perform the matrix multiplication
      nvcuda::wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }

    nvcuda::wmma::store_matrix_sync(&cS[(j + (i * scalingFactor)) * tDim * tDim], c_frag, tDim, nvcuda::wmma::mem_row_major);

    __syncthreads();
  }

  __syncthreads();

  // store c
  for (int o = 0; o < sDim; o += oFactor) {
    const int maxBeats = sDim / warpSize;

    for (int beat = 0; beat < maxBeats; beat++) {
      const int jIdx = jWT + (beat * warpSize);
      const int iIdx = jW + o;

      const int jIdxInBlock = jIdx % tDim;
      const int jBlockIdx = jIdx / tDim;
      const int iIdxInBlock = iIdx % tDim;
      const int iBlockIdx = iIdx / tDim;

      const int blockIdx = jBlockIdx + (iBlockIdx * scalingFactor);

      c[jIdx + (jB * sDim) + ((iIdx + (iB * sDim)) * n)] = cS[jIdxInBlock + (iIdxInBlock * tDim) + (blockIdx * tDim * tDim)];
    }
  }
}

static inline void mat_mulT(const half* da, const half* db, half* dc,
                            size_t m, size_t n, size_t k) {

  // allocating 16 warps
  // 1 warp per 1 tDim*tDim matrix
  constexpr int threads = 512;
  constexpr int scalingFactor = 4;

  // the kernel parameter
  constexpr int tDim = 16;

  // this logic is also present in the mat_mul_dt
  constexpr int sDim = tDim * scalingFactor;

  int blocksY = (m + sDim - 1) / sDim;
  int blocksX = (n + sDim - 1) / sDim;

  // all threads will be in the X dimension
  // warp level scheduling is performed internal to kernel
  dim3 threadsDim(threads);
  dim3 blocksDim(blocksX, blocksY);

  // printf("blocksX: %d, blocksY: %d\n", blocksX, blocksY);

  mat_mul_dt<threads, scalingFactor><<<blocksDim, threadsDim>>>(da, db, dc, m, n, k);
  expErrChk(hipGetLastError());
}

template<typename T>
static void print_mat(T *a, size_t m, size_t n) {
  for (size_t i = 0; i < m; i++) {
    for (size_t j = 0; j < n; j++) {
      printf("%.2f, ", (float) a[j + i * n]);
    }
    printf("\n");
  }
}

static float time_micro() {
  // DO NOT use chrono, it's just not precise enough

  struct timespec ts;
  clock_gettime(CLOCK_MONOTONIC_RAW, &ts);

  float time = (ts.tv_sec * 1000000.0f) + (ts.tv_nsec / 1000.0f);

  return time;
}

int main() {
  constexpr int reps = 10;

  // srand((unsigned int) time_micro());
  srand(0);

  printf("rand: %f\n", randf());
  printf("time: %f\n", time_micro());

  size_t size = 8192;
  size_t m = size;
  size_t n = size;
  size_t k = size;

  // a -> m * k
  // b -> k * n
  // c -> m * n
  float *a = (float *) malloc(sizeof(float) * m * k);
  float *b = (float *) malloc(sizeof(float) * k * n);
  float *cS = (float *) malloc(sizeof(float) * m * n);
  float *c_ref = (float *) malloc(sizeof(float) * m * n);
  half *aH = (half *) malloc(sizeof(half) * m * k);
  half *bH = (half *) malloc(sizeof(half) * k * n);
  half *cH = (half *) malloc(sizeof(half) * m * n);
  half *cHcuBlas = (half *) malloc(sizeof(half) * m * n);

  if (NULL == a || NULL == b || NULL == cS) {
    printf("Error allocating\n");
    goto end;
  }

  if (NULL == aH || NULL == bH || NULL == cH || NULL == cHcuBlas) {
    printf("Error allocating\n");
    goto end;
  }

  init_mat(a, m, k);
  copy_mat(aH, a, m * k);
  init_mat(b, k, n);
  copy_mat(bH, b, k * n);

  float *da;
  float *db;
  float *dc;

  printf("CUDA incepted\n");

  expErrChk(hipMalloc((void **) &da, sizeof(float) * m * k));
  expErrChk(hipMalloc((void **) &db, sizeof(float) * k * n));
  expErrChk(hipMalloc((void **) &dc, sizeof(float) * m * n));

  /*
  {
    // this takes long enough
    // skipping with one rep
    const int reps = 1;

    printf("CUDA starting\n");

    expErrChk(cudaMemcpy(da, a, sizeof(float) * m * k, cudaMemcpyHostToDevice));
    expErrChk(cudaMemcpy(db, b, sizeof(float) * k * n, cudaMemcpyHostToDevice));
    expErrChk(cudaMemset(dc, 0, sizeof(float) * m * n));

    const float begin = time_micro();

    for (int rep = 0; rep < reps; rep++) {
      mat_mul(da, db, dc, m, n, k);
    }

    const float end = time_micro();

    expErrChk(cudaMemcpy(cS, dc, sizeof(float) * m * n, cudaMemcpyDeviceToHost));

    printf("CUDA complete, end: %f, begin: %f, time/rep = %f\n", end, begin, ((end - begin) / reps));
  }
  */

  expErrChk(hipFree(dc));
  expErrChk(hipFree(db));
  expErrChk(hipFree(da));

  printf("Tensor CUDA incepted\n");

  half *daH;
  half *dbH;
  half *dcH;

  expErrChk(hipMalloc((void **) &daH, sizeof(half) * m * k));
  expErrChk(hipMalloc((void **) &dbH, sizeof(half) * k * n));
  expErrChk(hipMalloc((void **) &dcH, sizeof(half) * m * n));

  expErrChk(hipMemcpy(daH, aH, sizeof(half) * m * k, hipMemcpyHostToDevice));
  expErrChk(hipMemcpy(dbH, bH, sizeof(half) * k * n, hipMemcpyHostToDevice));

  {
    printf("Tensor CUDA Starting\n");
    expErrChk(hipMemset(dcH, 0, sizeof(half) * m * n));

    const float begin = time_micro();

    for (int rep = 0; rep < reps; rep++) {
      printf("rep: %d\n", rep);
      mat_mulT(daH, dbH, dcH, m, n, k);
      expErrChk(hipDeviceSynchronize());
    }

    const float end = time_micro();

    expErrChk(hipMemcpy(cH, dcH, sizeof(half) * m * n, hipMemcpyDeviceToHost));

    printf("Tensor CUDA complete, end: %f, begin: %f, time/rep = %f\n", end, begin, ((end - begin) / reps));
  }

  expErrChk(hipFree(dcH));

  printf("cuBLAS incepted\n");

  half* dcHcuBlas;
  expErrChk(hipMalloc((void **) &dcHcuBlas, sizeof(half) * m * n));

  {
    hipblasHandle_t handle;
    expErrChk(hipblasCreate(&handle));
    expErrChk(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

    const half alpha(1.0f);
    const half beta(1.0f);

    printf("cuBLAS Starting\n");
    expErrChk(hipMemset(dcHcuBlas, 0, sizeof(half) * m * n));

    const float begin = time_micro();

    for (int rep = 0; rep < reps; rep++) {
      printf("rep: %d\n", rep);
      expErrChk(hipblasHgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, k, &alpha, daH, k, dbH, n, &beta, dcHcuBlas, n));
      expErrChk(hipDeviceSynchronize());
    }

    const float end = time_micro();

    expErrChk(hipMemcpy(cHcuBlas, dcHcuBlas, sizeof(half) * m * n, hipMemcpyDeviceToHost));
    expErrChk(hipblasDestroy(handle));

    printf("cuBLAS complete, end: %f, begin: %f, time/rep = %f\n", end, begin, ((end - begin) / reps));
  }

  expErrChk(hipFree(dcHcuBlas));
  expErrChk(hipFree(dbH));
  expErrChk(hipFree(daH));

  // printf("cH:\n");
  // print_mat(cH, m, n);
  // printf("-------------------------------------------------\n");

  /*
  printf("cS:\n");
  print_mat(cS, m, n);
  printf("-------------------------------------------------\n");
  printf("cH:\n");
  print_mat(cH, m, n);
  printf("-------------------------------------------------\n");

  assert(reps == 1);

  const size_t m_block = 32;
  const size_t n_block = 32;
  const size_t k_block = 32;

  #pragma omp parallel for
  for (size_t io = 0; io < m; io += m_block) {
    for (size_t jo = 0; jo < n; jo += n_block) {
      for (size_t po = 0; po < k; po += k_block) {

        for (size_t ii = 0; ii < m_block; ii++) {
          for (size_t ji = 0; ji < n_block; ji++) {
            size_t i = io + ii;
            size_t j = jo + ji;

            float sum = c_ref[i * n + j];

            for (size_t pi = 0; pi < k_block; pi++) {
              size_t p = po + pi;

              sum += a[i * k + p] * b[p * k + j];
            }

            c_ref[i * n + j] = sum;
          }
        }
      }
    }
  }

  printf("Comparing CUDA\n");
  for (size_t i = 0; i < m * n; i++) {
    float c = cS[i];
    float sum = abs(c + c_ref[i]);
    float diff = abs(c - c_ref[i]);

    sum = (sum == 0) ? 1 : sum;

    if (diff > 1e-3 && (diff / sum) > 1e-3) {
      printf("Error at idx: %zu, result: %f, expected: %f\n", i, c, c_ref[i]);
      break;
    }
  }
  */

  printf("Comparing Tensors CUDA\n");
  // half has really low precision
  // any reps more than 1 might throw off the
  // comparators
  for (size_t i = 0; i < m; i++) {
    for (size_t j = 0; j < n; j++) {
      float c = cH[i * n + j];
      float ac = cHcuBlas[j * m + i];

      float sum = abs(c + ac);
      float diff = abs(c - ac);

      sum = (sum == 0) ? 1 : sum;

      if (diff > 1e-3 && (diff / sum) > 1e-2) {
        printf("Error at idx: %zu, %zu, result: %f, expected: %f\n", i, j, c, ac);
        goto end;
      }
    }
  }

  printf("Completed\n");

  end:
  if (cHcuBlas) {
    free(cHcuBlas);
  }

  if (cH) {
    free(cH);
  }

  if (bH) {
    free(bH);
  }

  if (aH) {
    free(aH);
  }

  if (cS) {
    free(cS);
  }

  if (b) {
    free(b);
  }

  if (a) {
    free(a);
  }

  return 0;
}
